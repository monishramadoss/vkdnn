#include "hip/hip_runtime.h"
#pragma once

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))
const int WARPSIZE = 32; // warpSize is not constexpr

template <const int BM, const int BN, const int BK, const int rowStrideA, const int rowStrideB>
__device__ void loadFromGmem(int N, int K, const float *A, const float *B, float *As, float *Bs,
                             int innerRowA, int innerColA, int innerRowB, int innerColB)
{
    for (uint offset = 0; offset + rowStrideA <= BM; offset += rowStrideA)
    {
        const float4 tmp = reinterpret_cast<const float4 *>(
            &A[(innerRowA + offset) * K + innerColA * 4])[0];
        // float4 tmp;
        // asm("ld.global.nc.v4.f32 {%0, %1, %2, %3}, [%4];"
        //     : "=f"(tmp.x), "=f"(tmp.y), "=f"(tmp.z), "=f"(tmp.w)
        //     : "l"(&A[(innerRowA + offset) * K + innerColA * 4]));
        As[(innerColA * 4 + 0) * BM + innerRowA + offset] = tmp.x;
        As[(innerColA * 4 + 1) * BM + innerRowA + offset] = tmp.y;
        As[(innerColA * 4 + 2) * BM + innerRowA + offset] = tmp.z;
        As[(innerColA * 4 + 3) * BM + innerRowA + offset] = tmp.w;
    }

    for (uint offset = 0; offset + rowStrideB <= BK; offset += rowStrideB)
    {
        reinterpret_cast<float4 *>(
            &Bs[(innerRowB + offset) * BN + innerColB * 4])[0] =
            reinterpret_cast<const float4 *>(
                &B[(innerRowB + offset) * N + innerColB * 4])[0];
        // asm("ld.global.v4.f32 {%0, %1, %2, %3}, [%4];"
        //     : "=f"(Bs[(innerRowB + offset) * BN + innerColB * 4 + 0]),
        //       "=f"(Bs[(innerRowB + offset) * BN + innerColB * 4 + 1]),
        //       "=f"(Bs[(innerRowB + offset) * BN + innerColB * 4 + 2]),
        //       "=f"(Bs[(innerRowB + offset) * BN + innerColB * 4 + 3])
        //     : "l"(&B[(innerRowB + offset) * N + innerColB * 4]));
    }
}

template <const int BM, const int BN, const int BK, const int WM, const int WN,
          const int WMITER, const int WNITER, const int WSUBM, const int WSUBN,
          const int TM, const int TN>
__device__ void
processFromSmem(float *regM, float *regN, float *threadResults, const float *As, const float *Bs, const uint warpRow, const uint warpCol,
                const uint threadRowInWarp, const uint threadColInWarp)
{
    for (uint dotIdx = 0; dotIdx < BK; ++dotIdx)
    {
        // populate registers for whole warptile
        for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx)
        {
            for (uint i = 0; i < TM; ++i)
            {
                regM[wSubRowIdx * TM + i] =
                    As[(dotIdx * BM) + warpRow * WM + wSubRowIdx * WSUBM +
                       threadRowInWarp * TM + i];
            }
        }
        for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx)
        {
            for (uint i = 0; i < TN; ++i)
            {
                regN[wSubColIdx * TN + i] =
                    Bs[(dotIdx * BN) + warpCol * WN + wSubColIdx * WSUBN +
                       threadColInWarp * TN + i];
            }
        }

        // execute warptile matmul
        for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx)
        {
            for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx)
            {
                // calculate per-thread results
                for (uint resIdxM = 0; resIdxM < TM; ++resIdxM)
                {
                    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN)
                    {
                        threadResults[(wSubRowIdx * TM + resIdxM) * (WNITER * TN) + (wSubColIdx * TN) + resIdxN] +=
                            regM[wSubRowIdx * TM + resIdxM] *
                            regN[wSubColIdx * TN + resIdxN];
                    }
                }
            }
        }
    }
}

// namespace wt

/*
 * @tparam BM The threadblock size for M dimension SMEM caching.
 * @tparam BN The threadblock size for N dimension SMEM caching.
 * @tparam BK The threadblock size for K dimension SMEM caching.
 * @tparam WM M dim of continuous tile computed by each warp
 * @tparam WN N dim of continuous tile computed by each warp
 * @tparam WMITER The number of subwarp tiling steps in M dimension.
 * @tparam WNITER The number of subwarp tiling steps in N dimension.
 * @tparam TM The per-thread tile size for M dimension.
 * @tparam TN The per-thread tile size for N dimension.
 */
template <const int BM, const int BN, const int BK, const int WM, const int WN,
          const int WNITER, const int TM, const int TN, const int NUM_THREADS>
__global__ void __launch_bounds__(NUM_THREADS)
    sgemmWarptiling(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    const uint cRow = blockIdx.y;
    const uint cCol = blockIdx.x;

    // Placement of the warp in the threadblock tile
    const uint warpIdx = threadIdx.x / WARPSIZE; // the warp this thread is in
    const uint warpCol = warpIdx % (BN / WN);
    const uint warpRow = warpIdx / (BN / WN);

    // size of the warp subtile
    constexpr uint WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
    constexpr uint WSUBM = WM / WMITER; // 64/2=32
    constexpr uint WSUBN = WN / WNITER; // 32/2=16

    // Placement of the thread in the warp subtile
    const uint threadIdxInWarp = threadIdx.x % WARPSIZE;         // [0, 31]
    const uint threadColInWarp = threadIdxInWarp % (WSUBN / TN); // i%(16/4)
    const uint threadRowInWarp = threadIdxInWarp / (WSUBN / TN); // i/4

    // allocate space for the current blocktile in SMEM
    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

    // Move blocktile to beginning of A's row and B's column
    A += cRow * BM * K;
    B += cCol * BN;
    // Move C_ptr to warp's output tile
    C += (cRow * BM + warpRow * WM) * N + cCol * BN + warpCol * WN;

    // calculating the indices that this thread will load into SMEM
    // we'll load 128bit / 32bit = 4 elements per thread at each step
    const uint innerRowA = threadIdx.x / (BK / 4);
    const uint innerColA = threadIdx.x % (BK / 4);
    constexpr uint rowStrideA = (NUM_THREADS * 4) / BK;
    const uint innerRowB = threadIdx.x / (BN / 4);
    const uint innerColB = threadIdx.x % (BN / 4);
    constexpr uint rowStrideB = NUM_THREADS / (BN / 4);

    // allocate thread-local cache for results in registerfile
    float threadResults[WMITER * TM * WNITER * TN] = {0.0};
    // we cache into registers on the warptile level
    float regM[WMITER * TM] = {0.0};
    float regN[WNITER * TN] = {0.0};

    // outer-most loop over block tiles
    for (uint bkIdx = 0; bkIdx < K; bkIdx += BK)
    {
        loadFromGmem<BM, BN, BK, rowStrideA, rowStrideB>(N, K, A, B, As, Bs, innerRowA, innerColA, innerRowB, innerColB);
        __syncthreads();

        processFromSmem<BM, BN, BK, WM, WN, WMITER, WNITER, WSUBM, WSUBN, TM, TN>(regM, regN, threadResults, As, Bs, warpRow, warpCol, threadRowInWarp, threadColInWarp);

        A += BK;     // move BK columns to right
        B += BK * N; // move BK rows down
        __syncthreads();
    }

    // write out the results
    for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx)
    {
        for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx)
        {
            // move C pointer to current warp subtile
            float *C_interim = C + (wSubRowIdx * WSUBM) * N + wSubColIdx * WSUBN;
            for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1)
            {
                for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4)
                {
                    // load C vector into registers
                    float4 tmp = reinterpret_cast<float4 *>(
                        &C_interim[(threadRowInWarp * TM + resIdxM) * N +
                                   threadColInWarp * TN + resIdxN])[0];
                    // perform GEMM update in reg
                    const int i = (wSubRowIdx * TM + resIdxM) * (WNITER * TN) +
                                  wSubColIdx * TN + resIdxN;
                    tmp.x = alpha * threadResults[i + 0] + beta * tmp.x;
                    tmp.y = alpha * threadResults[i + 1] + beta * tmp.y;
                    tmp.z = alpha * threadResults[i + 2] + beta * tmp.z;
                    tmp.w = alpha * threadResults[i + 3] + beta * tmp.w;
                    // write back
                    reinterpret_cast<float4 *>(
                        &C_interim[(threadRowInWarp * TM + resIdxM) * N +
                                   threadColInWarp * TN + resIdxN])[0] = tmp;
                }
            }
        }
    }
}

void runSgemmWarptiling(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    const uint K10_NUM_THREADS = 128;
    const uint K10_BN = 128;
    const uint K10_BM = 128;
    const uint K10_BK = 16;
    const uint K10_WN = 64;
    const uint K10_WM = 64;
    const uint K10_TN = 4;
    const uint K10_TM = 8;
    const uint K10_WNITER = 4;
    constexpr uint K10_WMITER = (K10_WM * K10_WN) / (32 * K10_TM * K10_TN * K10_WNITER);

    constexpr uint NUM_WARPS = K10_NUM_THREADS / 32;

    dim3 blockDim(K10_NUM_THREADS);
    dim3 gridDim(CEIL_DIV(N, K10_BN), CEIL_DIV(M, K10_BM));

    sgemmWarptiling<K10_BM, K10_BN, K10_BK, K10_WM, K10_WN, K10_WNITER, K10_TM, K10_TN, K10_NUM_THREADS>
        <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void range_init_matrix(float *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = i;
    }
}

void one_init_matrix(float *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = 1.0;
    }
}

void zero_init_matrix(float *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = 0.0;
    }
}

void copy_matrix(const float *src, float *dest, int N)
{
    int i;
    for (i = 0; src + i && dest + i && i < N; i++)
        *(dest + i) = *(src + i);
    if (i != N)
        printf("copy failed at %d while there are %d elements in total.\n", i, N);
}


#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>

void main()
{

    // cuBLAS FLOPs ceiling is reached at 8192
    std::vector<int> SIZE = {128, 256, 512, 1024, 2048, 4096};

    long m, n, k, max_size;
    max_size = SIZE[SIZE.size() - 1];
    float alpha = 0.5, beta = 3.0; // GEMM input parameters, C=α*AB+β*C

    float *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr; // host matrices
    float *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr; // device matrices

    A = (float *)malloc(sizeof(float) * max_size * max_size);
    B = (float *)malloc(sizeof(float) * max_size * max_size);
    C = (float *)malloc(sizeof(float) * max_size * max_size);
    C_ref = (float *)malloc(sizeof(float) * max_size * max_size);

    hipMalloc((void **)&dA, sizeof(float) * max_size * max_size);
    hipMalloc((void **)&dB, sizeof(float) * max_size * max_size);
    hipMalloc((void **)&dC, sizeof(float) * max_size * max_size);
    hipMalloc((void **)&dC_ref, sizeof(float) * max_size * max_size);

    one_init_matrix(A, max_size * max_size);
    one_init_matrix(B, max_size * max_size);
    zero_init_matrix(C, max_size * max_size);
    zero_init_matrix(C_ref, max_size * max_size);

    

}